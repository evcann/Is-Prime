#include "hip/hip_runtime.h"
#include <hiprand/hiprand_kernel.h>
#include ""
#include <stdio.h>
#include <hip/hip_runtime.h>

__global__ void is_prime(int* input, bool* output, int size) {
    int id = blockIdx.x * blockDim.x + threadIdx.x;
    if (id < size) {
        output[id] = false;
    }
}

int main() {
    const int size = 10;
    int input[] = { 1, 2, 3, 4, 5, 6, 7, 8, 9, 10 };
    bool output[size];

    int* d_input;
    bool* d_output;
    hipMalloc(&d_input, size * sizeof(int));
    hipMalloc(&d_output, size * sizeof(bool));
    hipMemcpy(d_input, input, size * sizeof(int), hipMemcpyHostToDevice);

    is_prime << <1000, 256>> > (d_input, d_output, size);

    hipMemcpy(output, d_output, size * sizeof(bool), hipMemcpyDeviceToHost);

    hipFree(d_input);
    hipFree(d_output);
    return 0;
}